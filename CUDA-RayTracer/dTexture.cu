#include "hip/hip_runtime.h"
/*
 Note to future self.

 This code is really bad. The only "Good" function here is load_texture_uchar. You need to ensure
 that data is properly cleaned up after use. Also, ensure that the mipmappedArray is cleaned up before returning.

 The best thing to do here is to make a class that handels all this bullshit. I'm just too lazy to do that atm.

*/

#include "dTexture.h"
#include "Texture.h"
#include "CudaHelpers.h"
#include <stb_image.h>

uint32_t get_mip_map_levels(hipExtent size)
{
    size_t sz = MAX(MAX(size.width, size.height), size.depth);

    uint32_t levels = 0;

    while (sz)
    {
        sz /= 2;
        levels++;
    }

    return levels;
}

hipTextureObject_t load_texture_uchar(Texture* tex)
{
    // TODO: make a dTexture class. This code is not great :( Need a better way of handling different texture formats/internal formats/dimensions.
    if (tex != NULL) {
        hipTextureObject_t textureObject;
        hipMipmappedArray_t mipmapArray;

        auto size = make_hipExtent(tex->width, tex->height, 0);
        uint32_t levels = get_mip_map_levels(size);

        hipChannelFormatDesc desc;
        size_t pitch;

        if (tex->nrComponents == 3) {
            pitch = size.width * sizeof(uchar4);
            desc = hipCreateChannelDesc<uchar4>();
            checkCudaErrors(hipMallocMipmappedArray(&mipmapArray, &desc, size, levels));

            unsigned char* data = (unsigned char*)malloc(tex->width * tex->height * sizeof(uchar4));

            int i = 0;
            int j = 0;
            while (i < tex->width * tex->height * 4) {
                data[i] = tex->data[j];
                data[i + 1] = tex->data[j + 1];
                data[i + 2] = tex->data[j + 2];
                data[i + 3] = 0;

                i += 4;
                j += 3;
            }
            hipArray_t level0;
            checkCudaErrors(hipGetMipmappedArrayLevel(&level0, mipmapArray, 0));

            hipMemcpy3DParms copyParams = { 0 };
            copyParams.srcPtr = make_hipPitchedPtr(data, pitch, size.width, size.height);
            copyParams.dstArray = level0;
            copyParams.extent = size;
            copyParams.extent.depth = 1;
            copyParams.kind = hipMemcpyHostToDevice;
            checkCudaErrors(hipMemcpy3D(&copyParams));

            free(data);
        }
        else {
            
            pitch = size.width * sizeof(uchar4);
            desc = hipCreateChannelDesc<uchar4>();
            checkCudaErrors(hipMallocMipmappedArray(&mipmapArray, &desc, size, levels));
            
            hipArray_t level0;
            checkCudaErrors(hipGetMipmappedArrayLevel(&level0, mipmapArray, 0));

            hipMemcpy3DParms copyParams = { 0 };
            copyParams.srcPtr = make_hipPitchedPtr(tex->data, pitch, size.width, size.height);
            copyParams.dstArray = level0;
            copyParams.extent = size;
            copyParams.extent.depth = 1;
            copyParams.kind = hipMemcpyHostToDevice;
            checkCudaErrors(hipMemcpy3D(&copyParams));
        }

        // compute rest of mipmaps based on level 0
        //generateMipMaps(mipmapArray, size);

        // generate bindless texture object
        
        hipResourceDesc resDescr;
        memset(&resDescr, 0, sizeof(hipResourceDesc));

        resDescr.resType = hipResourceTypeMipmappedArray;
        resDescr.res.mipmap.mipmap = mipmapArray;

        hipTextureDesc texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));

        texDescr.normalizedCoords = 1;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.mipmapFilterMode = hipFilterModeLinear;

        texDescr.addressMode[0] = hipAddressModeWrap;
        texDescr.addressMode[1] = hipAddressModeWrap;
        texDescr.addressMode[2] = hipAddressModeWrap;

        texDescr.maxMipmapLevelClamp = float(levels - 1);

        texDescr.readMode = hipReadModeNormalizedFloat;

        checkCudaErrors(hipCreateTextureObject(&textureObject, &resDescr, &texDescr, NULL));
        //checkCudaErrors(hipFreeMipmappedArray(mipmapArray));

        return textureObject;
    }
    else {
        return -1;
    }
}

hipTextureObject_t load_texture_uchar(std::string filename)
{
    int width, height, nrComponents;
    stbi_set_flip_vertically_on_load(true);
    unsigned char* hData = stbi_load(filename.c_str(), &width, &height, &nrComponents, 0);
    stbi_set_flip_vertically_on_load(false);
    unsigned int size = width * height * sizeof(float4);

    if (hData != nullptr) {

        hipChannelFormatDesc desc;
        hipArray* cuArray;

        desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindFloat);

        checkCudaErrors(hipMallocArray(&cuArray,
            &desc,
            width,
            height));

        if (nrComponents == 3) {

            unsigned int* data = (unsigned int*)malloc(size);

            int i = 0;
            int j = 0;
            while (i < width * height * 4) {
                data[i] = hData[j];
                data[i + 1] = hData[j + 1];
                data[i + 2] = hData[j + 2];
                data[i + 3] = 0;

                i += 4;
                j += 3;
            }
            checkCudaErrors(hipMemcpyToArray(cuArray,
                0,
                0,
                data,
                size,
                hipMemcpyHostToDevice));
        }
        else {
            checkCudaErrors(hipMemcpyToArray(cuArray,
                0,
                0,
                hData,
                size,
                hipMemcpyHostToDevice));
        }

        hipTextureObject_t         tex;
        hipResourceDesc            texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));

        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = cuArray;

        hipTextureDesc             texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));

        texDescr.normalizedCoords = true;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeWrap;
        texDescr.addressMode[1] = hipAddressModeWrap;
        texDescr.readMode = hipReadModeElementType;

        checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
        return tex;
    }
    else {
        std::cerr << "failed to load the texture \"" << filename << "\"." << endl;
        return -1;
    }
}

hipTextureObject_t load_texture_float(std::string filename)
{
    int width, height, nrComponents;

    //stbi_set_flip_vertically_on_load(true);
    float* hData = stbi_loadf(filename.c_str(), &width, &height, &nrComponents, 0);

    unsigned int size = width * height * sizeof(float4);

    if (hData != nullptr) {

        hipChannelFormatDesc desc;
        hipArray* cuArray;

        desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

        checkCudaErrors(hipMallocArray(&cuArray,
            &desc,
            width,
            height));

        if (nrComponents == 3) {

            float* data = (float*)malloc(size);

            int i = 0;
            int j = 0;
            while (i < width * height * 4) {
                data[i] = hData[j];
                data[i + 1] = hData[j + 1];
                data[i + 2] = hData[j + 2];
                data[i + 3] = 0;

                i += 4;
                j += 3;
            }
            checkCudaErrors(hipMemcpyToArray(cuArray,
                0,
                0,
                data,
                size,
                hipMemcpyHostToDevice));
        }
        else {
            checkCudaErrors(hipMemcpyToArray(cuArray,
                0,
                0,
                hData,
                size,
                hipMemcpyHostToDevice));
        }

        hipTextureObject_t         tex;
        hipResourceDesc            texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));

        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = cuArray;

        hipTextureDesc             texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));

        texDescr.normalizedCoords = true;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeWrap;
        texDescr.addressMode[1] = hipAddressModeWrap;
        texDescr.readMode = hipReadModeElementType;

        checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

        stbi_set_flip_vertically_on_load(false);

        return tex;
    }
    else {
        std::cerr << "failed to load the texture \"" << filename << "\"." << endl;
        return -1;
    }
}

hipSurfaceObject_t create_surface_float(int width, int height, int nrComponents)
{
    unsigned int size;
    hipChannelFormatDesc desc;
    hipArray* cuArray;

    switch (nrComponents) {
    case 1:
        size = width * height * sizeof(float);
        desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        break;
    case 2:
        size = width * height * sizeof(float2);
        desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
        break;
    default:
        size = width * height * sizeof(float4);
        desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    }

    checkCudaErrors(hipMallocArray(&cuArray,
        &desc,
        width,
        height));

    hipSurfaceObject_t         surf;
    hipResourceDesc            surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));

    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = cuArray;

    checkCudaErrors(hipCreateSurfaceObject(&surf, &surfRes));
    return surf;
}

void destroy_texture(hipTextureObject_t texture) {
    //hipTexObjectDestroy(texture);
    if (texture != -1) hipDestroyTextureObject(texture);
}
void destroy_surface(hipSurfaceObject_t surface) {
    //hipTexObjectDestroy(texture);
    if (surface != -1) hipDestroySurfaceObject(surface);
}